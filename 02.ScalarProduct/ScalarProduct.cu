#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include <malloc.h>
#include <iostream>
#include <string>
#include <time.h>
#include <numeric> 

#include "hip/hip_runtime.h"
#include ""

#define THREADS_PER_BLOCK 512

#define CHECK(call) \
do { \
	if (hipSuccess != call) { \
		fprintf(stderr, ("CUDA ERROR! file: %s[%i] -> %s\n"), __FILE__, __LINE__, hipGetErrorString(call)); \
		exit(0); \
	} \
} while (0)

void checkResult(float *hostRef, float *gpuRef, const int N)
{
	double epsilon = 1.0E-8;
	bool match = 1;

	if (abs(hostRef - gpuRef) > epsilon)
	{
		match = 0;
		printf("Arrays do not match!\n");
		printf("host %5.2f gpu %5.2f", hostRef,
			gpuRef);
	}
	if (match) printf("Arrays match.\n\n");

	return;
}

__global__ void scalarProductOnDevice(float *A, float *B, float *C, const int N)
{
	// multiplication 
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int temp[THREADS_PER_BLOCK];
	if (i < N) temp[threadIdx.x] = A[i] * B[i];

	__syncthreads();

	if (0 == threadIdx.x) {
		int sum = 0;

		for (int i = 0; i < blockDim.x; i++)
			sum += temp[i];
		atomicAdd(C, sum);
	}

}


void scalarProductOnHost(float *A, float *B, float *C, const int N)
{
	float product = 0;
	for (int idx = 0; idx < N; idx++)
	{
		product += (A[idx] * B[idx]);
	}
	*C = product;
}

void initialData(float *ip, int size)
{
	// generate different seed for random number
	time_t t;
	srand((unsigned)time(&t));

	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}

	return;
}

int main(int argc, char **argv)
{
	printf("%s Starting...\n", argv[0]);

	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = 1 << 10;
	printf("Vector size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	// initialize data at host side
	initialData(h_A, nElem);
	initialData(h_B, nElem);
	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	// add vector at host side for result checks
	scalarProductOnHost(h_A, h_B, hostRef, nElem);


	// malloc device global memory
	float *d_A, *d_B, *d_C;
	CHECK(hipMalloc((float**)&d_A, nBytes));
	CHECK(hipMalloc((float**)&d_B, nBytes));
	CHECK(hipMalloc((float**)&d_C, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	//int iLen = 512;
	dim3 blockDim(THREADS_PER_BLOCK);
	dim3 gridDim((nElem + blockDim.x - 1) / blockDim.x);



	// --------KERNEL coalesced sum
	scalarProductOnDevice << < gridDim, blockDim >> > (d_A, d_B, d_C, nElem);
	CHECK(hipDeviceSynchronize());

	// check kernel error
	CHECK(hipGetLastError());

	// copy kernel result back to host side
	CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

	// check device results
	checkResult(hostRef, gpuRef, nElem);

	// free device global memory
	CHECK(hipFree(d_A));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_C));

	// free host memory
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	return(0);
}