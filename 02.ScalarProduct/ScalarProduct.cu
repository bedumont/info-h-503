#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include <malloc.h>
#include <iostream>
#include <string>
#include <time.h>
#include <numeric> 

#include "hip/hip_runtime.h"
#include ""

#define CHECK(call) \
do { \
	if (hipSuccess != call) { \
		fprintf(stderr, ("CUDA ERROR! file: %s[%i] -> %s\n"), __FILE__, __LINE__, hipGetErrorString(call)); \
		exit(0); \
	} \
} while (0)

void checkResult(float *hostRef, float *gpuRef, const int N)
{
	double epsilon = 1.0E-8;
	bool match = 1;

	for (int i = 0; i < N; i++)
	{
		if (abs(hostRef[i] - gpuRef[i]) > epsilon)
		{
			match = 0;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
				gpuRef[i], i);
			break;
		}
	}

	if (match) printf("Arrays match.\n\n");

	return;
}

__global__ void scalarProductOnDevice(float *A, float *B, float *C, const int N)
{
	// multiplication 
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int temp[blockDim.x];
	if (i < N) temp[i] = A[i] * B[i];

	__syncthreads();

	if (0 == threadIdx.x) {
		int sum = 0;

		for( int i = O; i < blockDim.x ; i++)
	}

}


void scalarProductOnHost(float *A, float *B, float *C, const int N)
{
	float product = 0;
	for (int idx = 0; idx < N; idx++)
	{
		product += A[idx] * B[idx];
	}
}

int main(int argc, char **argv)
{
	printf("%s Starting...\n", argv[0]);

	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = 1 << 2;
	printf("Vector size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	//// initialize data at host side
	//initialData(h_A, nElem);
	//initialData(h_B, nElem);
	//memset(hostRef, 0, nBytes);
	//memset(gpuRef, 0, nBytes);

	//// add vector at host side for result checks
	//sumArraysOnHost(h_A, h_B, hostRef, nElem);


	//// malloc device global memory
	//float *d_A, *d_B, *d_C;
	//CHECK(hipMalloc((float**)&d_A, nBytes));
	//CHECK(hipMalloc((float**)&d_B, nBytes));
	//CHECK(hipMalloc((float**)&d_C, nBytes));

	//// transfer data from host to device
	//CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
	//CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
	//CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

	//// invoke kernel at host side
	//int iLen = 512;
	//dim3 blockDim(iLen);
	//dim3 gridDim((nElem + blockDim.x - 1) / blockDim.x);



	//// --------KERNEL coalesced sum
	//sumArraysOnGPU << <gridDim, blockDim >> > (d_A, d_B, d_C, nElem);
	//CHECK(hipDeviceSynchronize());

	//// check kernel error
	//CHECK(hipGetLastError());

	//// copy kernel result back to host side
	//CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

	//// check device results
	//checkResult(hostRef, gpuRef, nElem);

	//// free device global memory
	//CHECK(hipFree(d_A));
	//CHECK(hipFree(d_B));
	//CHECK(hipFree(d_C));

	//// free host memory
	//free(h_A);
	//free(h_B);
	//free(hostRef);
	//free(gpuRef);

	return(0);
}