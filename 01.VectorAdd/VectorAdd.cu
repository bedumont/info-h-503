#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include <malloc.h>
#include <iostream>
#include <string>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

/*
* This example demonstrates a simple vector sum on the GPU and on the host.
* sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
* GPU. Only a single thread block is used in this small case, for simplicity.
* sumArraysOnHost sequentially iterates through vector elements on the host.
* This version of sumArrays adds host timers to measure GPU and CPU
* performance.
*/

#define CHECK(call) \
do { \
	if (hipSuccess != call) { \
		fprintf(stderr, ("CUDA ERROR! file: %s[%i] -> %s\n"), __FILE__, __LINE__, hipGetErrorString(call)); \
		exit(0); \
	} \
} while (0)

void checkResult(float *hostRef, float *gpuRef, const int N)
{
	double epsilon = 1.0E-8;
	bool match = 1;

	for (int i = 0; i < N; i++)
	{
		if (abs(hostRef[i] - gpuRef[i]) > epsilon)
		{
			match = 0;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
				gpuRef[i], i);
			break;
		}
	}

	if (match) printf("Arrays match.\n\n");

	return;
}

void initialData(float *ip, int size)
{
	// generate different seed for random number
	time_t t;
	srand((unsigned)time(&t));

	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}

	return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
	for (int idx = 0; idx < N; idx++)
	{
		C[idx] = A[idx] + B[idx];
	}
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N) C[i] = A[i] + B[i];
}

__global__ void sumArraysOnGPUDivergence(float *A, float *B, float *C, const int N)
{
	// Can you draw the execution time of a warp?
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N) {
		if (threadIdx.x == 0) {
			int a;
			for (int j = 0; j < 10; j++) a += j;
		}
		C[i] = A[i] + B[i];
	}
}

__global__ void sumArraysOnGPUOffset(float *A, float *B, float *C, const int N)
{
	// What is the implications on the memory fetching process?
	// /!\ the sum of the vectors is not the right one. This kernel is here for the example
	// /!\ However the test will not fail if you don't change the offsets as C is already filled with right values
	const int offset = 10;
	const int offset2 = 5;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int j = i + offset;

	if (j < N) C[i + offset2] = A[j] + B[j];
}

__global__ void sumArraysOnGPUNoCoalescence(float *A, float *B, float *C, const int N)
{
	// Which thread is fetching which memory address?
	// /!\ the sum of the vectors is not the right one. This kernel is here for the example
	// /!\ However the test will not fail if you don't change the offsets as C is already filled with right values

	//int i = blockIdx.x * blockDim.x + threadIdx.x;
	int i = threadIdx.x * blockDim.x + blockIdx.x;

	if (i < N) C[i] = A[i] + B[i];
}

int main(int argc, char **argv)
{
	printf("%s Starting...\n", argv[0]);

	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = 1 << 10;
	printf("Vector size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	// initialize data at host side
	initialData(h_A, nElem);
	initialData(h_B, nElem);
	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	// add vector at host side for result checks
	sumArraysOnHost(h_A, h_B, hostRef, nElem);


	// malloc device global memory
	float *d_A, *d_B, *d_C;
	CHECK(hipMalloc((float**)&d_A, nBytes));
	CHECK(hipMalloc((float**)&d_B, nBytes));
	CHECK(hipMalloc((float**)&d_C, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 blockDim(iLen);
	dim3 gridDim((nElem + blockDim.x - 1) / blockDim.x);



	// --------KERNEL coalesced sum
	sumArraysOnGPU << <gridDim, blockDim >> >(d_A, d_B, d_C, nElem);
	CHECK(hipDeviceSynchronize());

	// check kernel error
	CHECK(hipGetLastError());

	// copy kernel result back to host side
	CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

	// check device results
	checkResult(hostRef, gpuRef, nElem);



	// --------KERNEL divergent sum
	sumArraysOnGPUDivergence << <gridDim, blockDim >> >(d_A, d_B, d_C, nElem);
	CHECK(hipDeviceSynchronize());

	// check kernel error
	CHECK(hipGetLastError());

	// copy kernel result back to host side
	CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

	// check device results
	checkResult(hostRef, gpuRef, nElem);



	// --------KERNEL coalesced sum with offset
	sumArraysOnGPUOffset << <gridDim, blockDim >> >(d_A, d_B, d_C, nElem);
	CHECK(hipDeviceSynchronize());

	// check kernel error
	CHECK(hipGetLastError());

	// copy kernel result back to host side
	CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

	// check device results
	checkResult(hostRef, gpuRef, nElem);



	// --------KERNEL non-coalesced sum
	sumArraysOnGPUNoCoalescence << <gridDim, blockDim >> >(d_A, d_B, d_C, nElem);
	CHECK(hipDeviceSynchronize());

	// check kernel error
	CHECK(hipGetLastError());

	// copy kernel result back to host side
	CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

	// check device results
	checkResult(hostRef, gpuRef, nElem);





	// free device global memory
	CHECK(hipFree(d_A));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_C));

	// free host memory
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	return(0);
}

