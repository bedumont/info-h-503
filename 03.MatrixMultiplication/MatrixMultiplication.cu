#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include <malloc.h>
#include <iostream>
#include <string>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#define CHECK(call) \
do { \
	if (hipSuccess != call) { \
		fprintf(stderr, ("CUDA ERROR! file: %s[%i] -> %s\n"), __FILE__, __LINE__, hipGetErrorString(call)); \
		exit(0); \
	} \
} while (0)

void checkResult(float *hostRef, float *gpuRef, const int Nx, const int Ny)
{
	double epsilon = 1.0E-8;
	bool match = 1;

	for (int i = 0; i < Nx*Ny; i++)
	{
		if (abs(hostRef[i] - gpuRef[i]) > epsilon)
		{
			match = 0;
			printf("Matrices do not match!\n");
			printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
				gpuRef[i], i);
			break;
		}
	}

	if (match) printf("Matrices match.\n\n");

	return;
}

void initialData(float *ip, int Nx, int Ny)
{
	// generate different seed for random number
	time_t t;
	srand((unsigned)time(&t));

	for (int i = 0; i < Nx*Ny; i++)
	{
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}

	return;
}

void multiplyMatricesOnHost(float *A, float *B, float *C, const int Nx, const int Ny)
{
	// TODO
	return;
}

__global__ void multiplyMatricesOnGPU(float *A, float *B, float *C, const int Nx, const int Ny)
{
	// TODO
	C[threadIdx.x] = 1.0f;
	return;
}

__global__ void multiplyMatricesOnGPUWithSharedMemory(float *A, float *B, float *C, const int Nx, const int Ny)
{
	// TODO
	C[threadIdx.x] = 1.0f;
	return;
}

int main(int argc, char **argv)
{
	printf("%s Starting...\n", argv[0]);

	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of matrices
	int nElemx = 1 << 16;
	int nElemy = 1 << 16;
	printf("Matrix size %dx%d\n", nElemx, nElemy);

	// malloc host memory
	size_t nBytes = nElemx*nElemy * sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	// initialize data at host side
	initialData(h_A, nElemx, nElemy);
	initialData(h_B, nElemx, nElemy);
	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	// Multiply matrices at host side for result checks
	multiplyMatricesOnHost(h_A, h_B, hostRef, nElemx, nElemy);


	// malloc device global memory
	float *d_A, *d_B, *d_C;
	CHECK(hipMalloc((float**)&d_A, nBytes));
	CHECK(hipMalloc((float**)&d_B, nBytes));
	CHECK(hipMalloc((float**)&d_C, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 blockDim(iLen);
	dim3 gridDim((nElemx*nElemy + blockDim.x - 1) / blockDim.x);

	// multiply kernel
	multiplyMatricesOnGPU << <gridDim, blockDim >> >(d_A, d_B, d_C, nElemx, nElemy);
	CHECK(hipDeviceSynchronize());

	// check kernel error
	CHECK(hipGetLastError());

	// copy kernel result back to host side
	CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

	// check device results
	checkResult(hostRef, gpuRef, nElemx, nElemy);

	// Cleanup GPU memor
	CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

	// multiply kernel with shared memory
	multiplyMatricesOnGPUWithSharedMemory << <gridDim, blockDim >> >(d_A, d_B, d_C, nElemx, nElemy);
	CHECK(hipDeviceSynchronize());

	// check kernel error
	CHECK(hipGetLastError());

	// copy kernel result back to host side
	CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

	// check device results
	checkResult(hostRef, gpuRef, nElemx, nElemy);



	// free device global memory
	CHECK(hipFree(d_A));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_C));

	// free host memory
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	return(0);
}

